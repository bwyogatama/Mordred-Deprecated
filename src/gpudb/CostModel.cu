#include "QueryOptimizer.h"
#include "CostModel.h"
#include "CPUGPUProcessing.h"

CostModel::CostModel(int _L, int _total_segment, int _n_group_key, int _n_aggr_key, int _sg, int _table_id, QueryOptimizer* _qo) {
	L = (double) _L;
	ori_L = (double) _L;
	n_group_key = _n_group_key;
	n_aggr_key = _n_aggr_key;

	sg = _sg;
	table_id = _table_id;

	qo = _qo;

	total_segment = _total_segment;

	Operator* op = qo->opRoots[table_id][sg];
	opPipeline.push_back(op);
	op = op->children;

	while (op != NULL) {
		if (op->type != Materialize && op->type != GPUtoCPU && op->type != CPUtoGPU && op->type != Merge) {
			opPipeline.push_back(op);
		}
		op = op->children;
	}
};

void 
CostModel::clear() {
	selectCPU.clear(); joinCPU.clear(); groupCPU.clear(); buildCPU.clear();
	selectGPU.clear(); joinGPU.clear(); groupGPU.clear(); buildGPU.clear();
}

void
CostModel::permute_cost() {
	for (int i = 0; i < opPipeline.size(); i++) {
		Operator* op = opPipeline[i];
		if (op->device == GPU) {
			if (op->type == Probe) joinGPU.push_back(op->columns[0]);
			else if (op->type == Filter) selectGPU.push_back(op->columns[0]);
			else if (op->type == GroupBy || op->type == Aggr) {
				for (int k = 0; k < op->columns.size(); k++)
					groupGPU.push_back(op->columns[k]);
			} else if (op->type == Build) buildGPU.push_back(op->columns[0]);
		} else {
			if (op->type == Probe) joinCPU.push_back(op->columns[0]);
			else if (op->type == Filter) selectCPU.push_back(op->columns[0]);
			else if (op->type == GroupBy || op->type == Aggr) {
				for (int k = 0; k < op->columns.size(); k++)
					groupCPU.push_back(op->columns[k]);
			} else if (op->type == Build) buildCPU.push_back(op->columns[0]);		
		}
	}

	double default_cost = calculate_cost();

	clear();

	for (int i = 0; i < opPipeline.size(); i++) {
		double cost = 0;

		Operator* cur_op = opPipeline[i];
		if (cur_op->device == CPU) cur_op->device = GPU;
		else if (cur_op->device == GPU) cur_op->device = CPU;

		for (int j = 0; j < opPipeline.size(); j++) {
			Operator* op = opPipeline[j];

			if (op->device == GPU) {
				if (op->type == Probe) joinGPU.push_back(op->columns[0]);
				else if (op->type == Filter) selectGPU.push_back(op->columns[0]);
				else if (op->type == GroupBy || op->type == Aggr) {
					for (int k = 0; k < op->columns.size(); k++)
						groupGPU.push_back(op->columns[k]);
				} else if (op->type == Build) buildGPU.push_back(op->columns[0]);	
			} else {
				if (op->type == Probe) joinCPU.push_back(op->columns[0]);
				else if (op->type == Filter) selectCPU.push_back(op->columns[0]);
				else if (op->type == GroupBy || op->type == Aggr) {
					for (int k = 0; k < op->columns.size(); k++)
						groupCPU.push_back(op->columns[k]);
				} else if (op->type == Build) buildCPU.push_back(op->columns[0]);			
			}
		}

		cost = calculate_cost();

		if (cur_op->device == CPU) {
			cur_op->device = GPU;
			for (int col = 0; col < cur_op->columns.size(); col++) {
				ColumnInfo* column = cur_op->columns[col];
				for (int seg = 0; seg < qo->segment_group_count[table_id][sg]; seg++) {
					int seg_id = qo->segment_group[table_id][sg * total_segment + seg];
					Segment* segment = qo->cm->index_to_segment[column->column_id][seg_id];
					qo->cm->updateSegmentWeightCostDirect(column, segment, (cost - default_cost) / qo->segment_group_count[table_id][sg] / cur_op->columns.size());
				}
			}
			for (int col = 0; col < cur_op->supporting_columns.size(); col++) {
				ColumnInfo* column = cur_op->supporting_columns[col];
				for (int seg_id = 0; seg_id < column->total_segment; seg_id++) {
					Segment* segment = qo->cm->index_to_segment[column->column_id][seg_id];
					qo->cm->updateSegmentWeightCostDirect(column, segment, (cost - default_cost) / column->total_segment);
				}
			}
		} else if (cur_op->device == GPU) {
			cur_op->device = CPU;
			for (int col = 0; col < cur_op->columns.size(); col++) {
				ColumnInfo* column = cur_op->columns[col];
				for (int seg = 0; seg < qo->segment_group_count[table_id][sg]; seg++) {
					int seg_id = qo->segment_group[table_id][sg * total_segment + seg];
					Segment* segment = qo->cm->index_to_segment[column->column_id][seg_id];
					qo->cm->updateSegmentWeightCostDirect(column, segment, (default_cost - cost) / qo->segment_group_count[table_id][sg]/ cur_op->columns.size());
				}
			}
			for (int col = 0; col < cur_op->supporting_columns.size(); col++) {
				ColumnInfo* column = cur_op->supporting_columns[col];
				for (int seg_id = 0; seg_id < column->total_segment; seg_id++) {
					Segment* segment = qo->cm->index_to_segment[column->column_id][seg_id];
					qo->cm->updateSegmentWeightCostDirect(column, segment, (default_cost - cost) / column->total_segment);
				}
			}
		}

		clear();
	}

}

double 
CostModel::calculate_cost() {
	double cost = 0;
	L = (double) ori_L;

	bool fromGPU = false;
	if (selectGPU.size() > 0 || joinGPU.size() > 0) {
		for (int i = 0; i < selectGPU.size(); ++i) {
			ColumnInfo* col = selectGPU[i];
			L *= qo->params->real_selectivity[col];
		}
		for (int i = 0; i < joinGPU.size(); ++i) {	
			ColumnInfo* col = joinGPU[i];
			L *= qo->params->real_selectivity[col];
		}
		if (selectCPU.size() > 0 || joinCPU.size() > 0 || groupCPU.size() > 0 || buildCPU.size() > 0) {
			cost += transfer_cost(joinGPU.size() + 1);
			fromGPU = true;
		}
	}

	for (int i = 0; i < selectCPU.size(); i++) {
		ColumnInfo* col = selectCPU[i];
		if (fromGPU) {
			cost += filter_cost(qo->params->real_selectivity[col], 1, 0);
			fromGPU = false;
		} else cost += filter_cost(qo->params->real_selectivity[col], 0, 0);
	}

	for (int i = 0; i < joinCPU.size(); i++) {
		ColumnInfo* col = joinCPU[i];
		if (fromGPU) {
			cost += probe_cost(qo->params->real_selectivity[col], 1, 0);
			fromGPU = false;
		} else cost += probe_cost(qo->params->real_selectivity[col], 0, 0);
	}

	if (groupCPU.size() > 0) {
		if (fromGPU){
			cost += group_cost(1);
			fromGPU = false;
		} else cost += group_cost(0);
	}

	//TODO: only works for SSB
	if (groupGPU.size() > 0 && (selectCPU.size() > 0 || joinCPU.size() > 0)) {
		// cost += transfer_cost(joinCPU.size() + joinGPU.size() + 1);
		cost += group_cost(0);
	} else if (groupGPU.size() > 0 && (selectGPU.size() > 0 || joinGPU.size() > 0)) {
		cost = 0;
	}

	//TODO: only works for SSB
	if (buildGPU.size() > 0 && (selectCPU.size() > 0 || joinCPU.size() > 0)) {
		cost += transfer_cost(joinCPU.size() + joinGPU.size() + 1);
	} else if (buildGPU.size() > 0 && (selectGPU.size() > 0 || joinGPU.size() > 0)) {
		cost = 0;
	}

	if (buildCPU.size() > 0) {
		if (fromGPU) {
			cost += build_cost(1);
			fromGPU = false;
		} else cost += build_cost(0);
	}

	return cost;

}

double 
CostModel::probe_cost(double selectivity, bool mat_start, bool mat_end) {

	double cost = 0;
	double scan_time = 0, probe_time = 0, write_time = 0;

	if (mat_start) scan_time = L * 4/BW_CPU + L * CACHE_LINE/BW_CPU;
	else scan_time = L * 4/BW_CPU;

	probe_time = L * CACHE_LINE/BW_CPU;

	if (mat_end) write_time = L * 4 * selectivity * 2/BW_CPU;
	else write_time = 0;

	L *= selectivity;

	cost = scan_time + probe_time + write_time;

	return cost;
}

double 
CostModel::transfer_cost(int M) {
	double transfer_time = L * 4 * M/BW_PCI;
	return transfer_time;
}

double 
CostModel::filter_cost(double selectivity, bool mat_start, bool mat_end) {

	double cost = 0;
	double scan_time = 0, write_time = 0;

	if (mat_start) scan_time = L * 4/BW_CPU + L * CACHE_LINE/BW_CPU;
	else scan_time = L * 4/BW_CPU;

	if (mat_end) write_time = L * 4 * selectivity/BW_CPU;
	else write_time = 0;

	L *= selectivity;

	cost = scan_time + write_time;

	return cost;
}

double 
CostModel::group_cost(bool mat_start) {

	double cost = 0;
	double scan_time = 0, group_time = 0;

	if (mat_start) scan_time = L * 4 /BW_CPU + L * CACHE_LINE * (n_aggr_key)/BW_CPU; //the cost to random read group key has not been included
	else scan_time = L * CACHE_LINE * n_aggr_key/BW_CPU;

	group_time = L * CACHE_LINE/BW_CPU;

	cost = scan_time + group_time;

	return cost;
}

double 
CostModel::build_cost(bool mat_start) {

	double cost = 0;
	double scan_time = 0, build_time = 0;

	if (mat_start) scan_time = L * 4/BW_CPU + L * CACHE_LINE/BW_CPU;
	else scan_time = L * 4/BW_CPU;

	build_time = L * CACHE_LINE/BW_CPU;

	cost = scan_time + build_time;

	return cost;
}